#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <stdlib.h>
#include <iterator> 

#include <png++/png.hpp>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "ca.cu"

// Random number in range functor
struct randomInts{

    private:
        const float a;
    
    public:
        randomInts(int A):a(A){}
    
        __host__ __device__
        int operator()(){ return (int)std::floor((float)(a)*rand()/RAND_MAX); }
};

// Print thrust vector values to console
template<class T>
void consolePrinter(T& x, std::string m, const char* seperator)
{
    std::cout << m;
    thrust::copy(x.begin(),x.end(),std::ostream_iterator<int>(std::cout,seperator));
    std::cout << "\n";
}

// Print values to grayscale png
template <class T,class I>
void pngPrinter(T& x,int n, I& image)
{   
    thrust::copy(x.begin(),x.end(),image[n].begin());
}

int main(int argc, char* argv[])
{    
    int range  = 3;              // Cell neighbour view range (default for now)
    int length = atoi(argv[1]); // Length of cell array
    int steps  = atoi(argv[2]);  // Number of update steps
    int states = atoi(argv[3]); // Number of states
    int rule   = atoi(argv[4]);   // Rule number
    
    int statePerms = pow(states,range);

    if (rule >=  pow(states,statePerms))
    {
        std::cout << "Rule outside range\n";
        return 999; 
    }

    // PNG image storage    
    png::image< png::gray_pixel > image(length,steps);

    // Generate random initial cell state 
    thrust::host_vector<int> init(length);
    thrust::generate(init.begin(),init.end(),randomInts(states));
    
    // Ruleset array
    thrust::host_vector<int> rules(statePerms);
    int x = rule;
    for(int i=0; i<statePerms; ++i)
    {
        rules[i] = x%(states);
        x = (int)floor((double)x/(double)states);
    }
    consolePrinter(rules,"Ruleset:|","|");
    
    // Load ruleset into device vector
    thrust::device_vector<int> d_rules(statePerms);
    thrust::copy(rules.begin(),rules.end(),d_rules.begin());

    // Colour transformation from states number to 8-bit grayscale
    thrust::device_vector<int> clr(length);
    thrust::fill(clr.begin(),clr.end(),255/(states-1));

    // Vector to copy to PNG image
    thrust::device_vector<int> outVec(length);

    // Initialize functor
    caUpdate CA(thrust::raw_pointer_cast(&d_rules[0]),states);
    
    // Time step counter
    int counter = 0;

    // Initialize CA object
    ca1d caa(length,states,CA);
    caa.loadInitial(init);

    while(counter < steps){

        // Update front array from back
        caa.updateFront();
        
        // Update colour value vector and copy to PNG
        thrust::transform(caa.ft.begin(),caa.ft.end(),clr.begin(),outVec.begin(),thrust::multiplies<int>());
        pngPrinter(outVec,counter,image);

        // Swap front and back, increment
        caa.swapFB();
        ++counter;
    }
    
    // Save PNG data
    image.write("pngs/rule_"+std::string(argv[4])+"_"+std::string(argv[3])+".png");

    return 0;
}