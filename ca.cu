#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/for_each.h>

struct randomInts{

    const float a;
    randomInts(int A):a(A){}
    __host__ __device__
    int operator()(){ return (int)std::floor((float)(a+1)*rand()/RAND_MAX); }
};

struct caUpdate{
    
    private:
    const int* ruleArr;

    public:
    caUpdate(int* x):ruleArr(x){}
    
    template <class Tuple>
    __device__
    void operator()(Tuple t)
    {
        int a = thrust::get<0>(t);
        int b = thrust::get<1>(t);
        int c = thrust::get<2>(t);
        thrust::get<3>(t) = *(ruleArr+b+2*a+4*c);
    }
};

void printer(thrust::host_vector<int>& x,std::string m){
    std::cout << m;
    thrust::copy(x.begin(),x.end(),std::ostream_iterator<int>(std::cout,"|"));
    std::cout << std::endl;
}

void printer(thrust::device_vector<int>& x,std::string m){
    std::cout << m;
    thrust::copy(x.begin(),x.end(),std::ostream_iterator<int>(std::cout,"|"));
    std::cout << std::endl;
}

int main(int argc, char* argv[]){

    // Host vector if random ints [0,range]
    int range = 1, length = atoi(argv[1]), rule=atoi(argv[2]), steps=atoi(argv(3));
    thrust::host_vector<int> init(length);
    thrust::generate(init.begin(),init.end(),randomInts(range));
    printer(init,"Initial state: ");
    
    // Ruleset matrix
    thrust::host_vector<int> rules(8);
    int x = rule;
    for(int i=0;i<8;++i)
    {
        rules[i] = x%(range+1);
        x = x >> 1;
    }
    std::cout << "Ruleset " << rule << ": [";
    thrust::copy(rules.begin(),rules.end(),std::ostream_iterator<int>(std::cout, "]["));
    std::cout << std::endl;

    // Maps to neighbours
    thrust::counting_iterator<int> it(0);
    thrust::device_vector<int> lft(length),rgt(length);
    lft[0] = length-1;
    thrust::copy(it,it+length,lft.begin()+1);
    thrust::copy(it+1,it+length,rgt.begin());
    rgt[length-1] = 0;
    
    thrust::device_vector<int> bk(length),ft(length),d_rules(8);
    thrust::copy(init.begin(),init.end(),bk.begin());
    thrust::copy(rules.begin(),rules.end(),d_rules.begin());

    caUpdate CA(thrust::raw_pointer_cast(&d_rules[0]));
    
    int counter = 0;

    while(counter < steps){

        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    bk.begin(),
                    thrust::make_permutation_iterator(bk.begin(),lft.begin()),
                    thrust::make_permutation_iterator(bk.begin(),rgt.begin()),
                    ft.begin()
                )
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    bk.end(),
                    thrust::make_permutation_iterator(bk.begin(),lft.end()),
                    thrust::make_permutation_iterator(bk.begin(),rgt.end()),
                    ft.end()
                )
            ),
            CA
        );

        printer(ft,"step: ");
        ft.swap(bk);
        ++counter;
    }
    
    

    return 0;
}