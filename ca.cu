#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <math.h>
#include <stdlib.h>
#include <iterator> 

#include <png++/png.hpp>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/for_each.h>

// Random number in range functor
struct randomInts{

    private:
    const float a;
    
    public:
    randomInts(int A):a(A){}
    
    __host__ __device__
    int operator()(){ return (int)std::floor((float)(a+1)*rand()/RAND_MAX); }
};

// CA Update process functor
struct caUpdate{
    
    private:
    const int* ruleArr;

    public:
    caUpdate(int* x):ruleArr(x){}
    
    template <class Tuple>
    __device__
    void operator()(Tuple t)
    {
        int a = thrust::get<0>(t);
        int b = thrust::get<1>(t);
        int c = thrust::get<2>(t);
        thrust::get<3>(t) = *(ruleArr+b+2*a+4*c);
    }
};

// Print thrust vector values to console
template<class T>
void consolePrinter(T& x, std::string m, const char* seperator)
{
    std::cout << m;
    thrust::copy(x.begin(),x.end(),std::ostream_iterator<int>(std::cout,seperator));
    std::cout << "\n";
}

// Print values to grayscale png
template <class T,class I>
void pngPrinter(T& x,int n, I& image)
{   
    thrust::copy(x.begin(),x.end(),image[n].begin());
}

int main(int argc, char* argv[])
{
    
    int range = 1;              // Cell neighbour view range
    int length = atoi(argv[1]); // Length of cell array
    int rule=atoi(argv[2]);     // Rule number
    int steps=atoi(argv[3]);    // Number of update steps

    // PNG image storage    
    png::image< png::gray_pixel > image(length,steps);

    // Generate random initial cell state 
    thrust::host_vector<int> init(length);
    thrust::generate(init.begin(),init.end(),randomInts(range));
    
    // Ruleset matrix
    thrust::host_vector<int> rules(8);
    int x = rule;
    for(int i=0;i<8;++i)
    {
        rules[i] = x%(range+1);
        x = x >> 1;
    }
    consolePrinter(rules,"Ruleset:|","|");
    
    // Maps to neighbouring cells
    thrust::counting_iterator<int> it(0);
    thrust::device_vector<int> lft(length),rgt(length);
    lft[0] = length-1;
    thrust::copy(it,it+length,lft.begin()+1);
    thrust::copy(it+1,it+length,rgt.begin());
    rgt[length-1] = 0;

    // Front, back and rulest device vectors
    thrust::device_vector<int> bk(length),ft(length),d_rules(8);
    thrust::copy(init.begin(),init.end(),bk.begin());
    thrust::copy(rules.begin(),rules.end(),d_rules.begin());

    // Colour transformation from states number to 8-bit grayscale
    thrust::device_vector<int> clr(length);
    thrust::fill(clr.begin(),clr.end(),255);

    // Vector to copy to PNG image
    thrust::device_vector<int> outVec(length);

    // Initialize functor
    caUpdate CA(thrust::raw_pointer_cast(&d_rules[0]));
    
    // Time step counter
    int counter = 0;

    while(counter < steps){

        // Zip vectors and perform update from back to front
        thrust::for_each(
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    bk.begin(),
                    thrust::make_permutation_iterator(bk.begin(),lft.begin()),
                    thrust::make_permutation_iterator(bk.begin(),rgt.begin()),
                    ft.begin()
                )
            ),
            thrust::make_zip_iterator(
                thrust::make_tuple(
                    bk.end(),
                    thrust::make_permutation_iterator(bk.begin(),lft.end()),
                    thrust::make_permutation_iterator(bk.begin(),rgt.end()),
                    ft.end()
                )
            ),
            CA
        );

        // Update colour value vector and copy to PNG
        thrust::transform(ft.begin(),ft.end(),clr.begin(),outVec.begin(),thrust::multiplies<int>());
        pngPrinter(outVec,counter,image);

        // Swap front and back, increment
        ft.swap(bk);
        ++counter;
    }
    
    // Save PNG data
    image.write("rule_"+std::string(argv[2])+".png");

    return 0;
}